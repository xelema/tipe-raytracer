#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "vec3.hu"
#include "ray.hu"
#include "hitinfo.hu"
#include "sphere.hu"
#include "rtutility.hu"
#include "camera.hu"

__constant__ const sphere sphere_list[10] = {
    {{{-501,0,0}}, 500, {GREEN, BLACK, 0.0}},                 
    // mur gauche vert
    {{{0,-501,0}}, 500, {WHITE, BLACK, 0.0}},                 
    // sol blanc
    {{{501, 0, 0}}, 500, {RED, BLACK, 0.0}},                  
    // mur droite rouge
    {{{-0.5, 1.4, -3}}, 0.5, {BLACK, {{1.0, 0.6, 0.2}}, 8.0}},   
    // LUMIERE (couleure noire, emission ORANGE)
    {{{0.5, 1.4, -3}}, 0.5, {BLACK, {{0.7, 0.2, 1.0}}, 8.0}},   
    // LUMIERE (couleure noire, emission VIOLETTE)
    {{{-0.5, -1.4, -3}}, 0.5, {BLACK, {{0.55, 0.863, 1.0}}, 5.0}},   
    // LUMIERE (couleure noire, emission CYAN)
    {{{0.5, -1.4, -3}}, 0.5, {BLACK, {{0.431, 1.0, 0.596}}, 5.0}},   
    // LUMIERE (couleure noire, emission VERT FLUO)
    {{{0, 0, -504}}, 500, {WHITE, BLACK, 0.0}},               
    // fond blanc
    {{{0, 501, 0}}, 500, {WHITE, BLACK, 0.0}},                
    // plafond blanc
    {{{0, 0, -3}}, 0.5, {SKY, BLACK, 0.0}}                    
    // boule bleue centrale (couleur ciel)
    };

///////////////////////////////////////////////////////////////////////////////////////////////

__device__ HitInfo hit_sphere(point3 center, double radius, ray r){

    HitInfo hitInfo;
    hitInfo.didHit=false; 

    //si delta>0 alors sphere il y a

    vec3 oc = sub(r.origin, center);
    double a = vec3_dot(r.dir, r.dir);
    double b = 2.0*vec3_dot(oc, r.dir);
    double c = vec3_dot(oc, oc) - radius*radius;
    
    double discriminant = b*b - 4*a*c;

    if (discriminant >= 0){
        double t1 = (-b -sqrt(discriminant))/(2*a);
        double t2 = (-b +sqrt(discriminant))/(2*a);
        
        if (t1 >= 0){
            hitInfo.didHit = true;
            hitInfo.dst = t1;
            hitInfo.hitPoint = ray_at(r, t1);
            hitInfo.normal = vec3_normalize(sub(ray_at(r, t1), center));
        }
        else if (t2 >= 0){
            hitInfo.didHit = true;
            hitInfo.dst = t2;
            hitInfo.hitPoint = ray_at(r, t2);
            hitInfo.normal = vec3_normalize(sub(ray_at(r, t2), center));
        }
    }
    return hitInfo;
}

__device__ HitInfo closest_hit(ray r){
    int nbSpheres = sizeof(sphere_list) / sizeof(sphere_list[0]);

    HitInfo closestHit;
    closestHit.didHit=false;
    closestHit.dst=INFINITY; // rien touché pour l'instant

    for(int i=0; i < nbSpheres ; i++){
        sphere s = sphere_list[i];
        HitInfo hitInfo =  hit_sphere(s.center, s.radius, r);

        if (hitInfo.didHit && hitInfo.dst < closestHit.dst){
            closestHit = hitInfo;
            closestHit.mat = s.mat;
        }
    }
    return closestHit;
}


__device__ point3 tracer(ray r, int nbRebondMax, hiprandState* globalState, int ind){

    color incomingLight = BLACK;
    color rayColor = WHITE;

    for (int i = 0; i<nbRebondMax; i++){

        HitInfo hitInfo = closest_hit(r);

        if (hitInfo.didHit){
            r.origin = hitInfo.hitPoint;
            r.dir = random_dir(hitInfo.normal, globalState, ind);

            material mat = hitInfo.mat;

            color emittedLight = multiply_scalar(mat.emissionColor, mat.emissionStrength);

            double lightStrength = vec3_dot(hitInfo.normal, r.dir); // Loi de Lambert

            incomingLight = add(incomingLight,multiply(emittedLight, rayColor));
            rayColor = multiply(multiply_scalar(mat.diffuseColor, lightStrength*2 /*trop sombre sinon*/ ), rayColor); 
        }
        else{
            break;
        }
    }
    return incomingLight;
}

__global__ void init_curand_state(hiprandState* states, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int ind = j * gridDim.x * blockDim.x + i;

    // chaque thread a le meme seed
    if (i < width && j < height)
        hiprand_init(6969, ind, 0, &states[ind]);
}

__global__ void render_kernel(color* canva, int image_width, int image_height, int nbRayonParPixel, int nbRebondMax, camera cam, hiprandState* states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int ind = j * gridDim.x * blockDim.x + i;

    if (i < image_width && j < image_height){
        int pixel_index = j*image_width+i;
        color totalLight = BLACK;
        
        for (int k=0; k<nbRayonParPixel; ++k){
            
            double u = ((double)i + randomDouble(states, ind, -0.5, 0.5))/(image_width-1);
            double v = ((double)j + randomDouble(states, ind, -0.5, 0.5))/(image_height-1);

            ray r = get_ray(u, v, cam);
            totalLight = add(totalLight, tracer(r, nbRebondMax, states, ind));
        }

        canva[pixel_index] = write_color_canva(totalLight, nbRayonParPixel);
    }
}

int main(){

    // CONSTANTES (paramètres de rendu)
    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////

    double ratio = 4.0 / 3.0;
    int largeur_image = 1200;
    int hauteur_image = (int)(largeur_image / ratio);

    int nbRayonParPixel = 1000;
    int nbRebondMax = 6;
    
    int nbThreadsX = 8; // peut dépendre des GPU
    int nbThreadsY = 8; 

    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////
    
    // temps d'execution
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // nom du fichier
    char nomFichier[100];
    time_t maintenant = time(NULL); // Obtenir l'heure actuelle
    struct tm *temps = localtime(&maintenant); // Convertir en structure tm

    sprintf(nomFichier, "CUDA_%dRAYS_%dRB_%02d-%02d_%02dh%02d.ppm", nbRayonParPixel, nbRebondMax-1, temps->tm_mday, temps->tm_mon + 1, temps->tm_hour, temps->tm_min);


    FILE *fichier = fopen(nomFichier, "w");

    // camera
    camera cam = init_camera(ratio);

    // tableau pour avoir chaque valeur de pixel au bon endroit (multithread et CUDA du coup)
    color* canva = (color*)malloc((largeur_image * hauteur_image)*sizeof(struct Vec3));
    for (int i = 0; i < largeur_image*hauteur_image; i++) {
        canva[i] = BLACK;
    }

    // alloue la mémoire pour canva sur le device (gpu)
    color* canva_device;
    hipMalloc((void**)&canva_device, (largeur_image * hauteur_image)*sizeof(color));

    // défini la taille des blocks et threads
    dim3 blocks(largeur_image/nbThreadsX+1, hauteur_image/nbThreadsY+1);
    dim3 threads(nbThreadsX, nbThreadsY);

    // alloue la mémoire pour states sur le device (gpu)
    hiprandState* states;
    hipMalloc((void**) &states, (largeur_image * hauteur_image) * sizeof(hiprandState));

    // initialise les "states" pour la fonction de random
    init_curand_state<<<blocks, threads>>>(states, largeur_image, hauteur_image);

    // lance le kernel avec le nb de thread défini
    render_kernel<<<blocks, threads>>>(canva_device, largeur_image, hauteur_image, nbRayonParPixel, nbRebondMax, cam, states);

    // copie le canva du device (gpu) vers l'host (cpu), puis free la mémoire du canva sur device
    hipMemcpy(canva, canva_device, (largeur_image * hauteur_image)*sizeof(color), hipMemcpyDeviceToHost);
    hipFree(canva_device);

    // base_ppm() et canva_to_ppm() réécrits ici
    fprintf(fichier, "P3\n%d %d\n255\n", largeur_image, hauteur_image);
    for (int j = hauteur_image-1; j >= 0  ; j--){ 
        for (int i = 0; i < largeur_image; i++){
            fprintf(fichier, "%d %d %d\n", (int)canva[j*largeur_image+i].e[0], (int)canva[j*largeur_image+i].e[1], (int)canva[j*largeur_image+i].e[2]);
        }
    }
    
    fclose(fichier);

    // enregistrer le moment d'arrivée
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    int minutes = (int)(elapsedTime / 60000);
    int seconds = (int)((elapsedTime - minutes * 60000) / 1000);
    
    fprintf(stderr, "\nFini.\n");
    fprintf(stderr, "Temps de rendu : %d min %d sec\n", minutes, seconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

	return 0;
}
