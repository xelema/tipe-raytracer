#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <OpenImageDenoise/oidn.h>

#include "vec3.hu"
#include "ray.hu"
#include "hitinfo.hu"
#include "sphere.hu"
#include "rtutility.hu"
#include "camera.hu"
#include "denoiser.hu"

__host__ __device__ HitInfo hit_sphere(point3 center, double radius, ray r){

    HitInfo hitInfo;
    hitInfo.didHit=false; 

    //si delta>0 alors spherse il y a

    vec3 oc = sub(r.origin, center);
    double a = vec3_dot(r.dir, r.dir);
    double b = 2.0*vec3_dot(oc, r.dir);
    double c = vec3_dot(oc, oc) - radius*radius;
    
    double discriminant = b*b - 4*a*c;

    if (discriminant >= 0){
        double t1 = (-b -sqrt(discriminant))/(2*a);
        double t2 = (-b +sqrt(discriminant))/(2*a);
        
        if (t1 >= 0){
            hitInfo.didHit = true;
            hitInfo.dst = t1;
            hitInfo.hitPoint = ray_at(r, t1);
            hitInfo.normal = vec3_normalize(sub(ray_at(r, t1), center));
        }
        else if (t2 >= 0){
            hitInfo.didHit = true;
            hitInfo.dst = t2;
            hitInfo.hitPoint = ray_at(r, t2);
            hitInfo.normal = vec3_normalize(sub(ray_at(r, t2), center));
        }
    }
    return hitInfo;
}

__host__ __device__ HitInfo closest_hit(ray r, sphere* spheres, int nbSpheres){

    HitInfo closestHit;
    closestHit.didHit=false;
    closestHit.dst=INFINITY; // rien touché pour l'instant

    for(int i=0; i < nbSpheres ; i++){
        sphere s = spheres[i];
        HitInfo hitInfo = hit_sphere(s.center, s.radius, r);

        if (hitInfo.didHit && hitInfo.dst < closestHit.dst){
            closestHit = hitInfo;
            closestHit.mat = s.mat;
        }
    }
    return closestHit;
}

__device__ color tracer(ray r, int nbRebondMax, hiprandState* globalState, int ind, sphere* spheres, int nbSpheres){

    HitInfo hitInfo = closest_hit(r, spheres, nbSpheres); // cas des lumières
    if (hitInfo.didHit){
        if (hitInfo.mat.emissionStrength > 0){
            color HSL = rgb_to_hsl(hitInfo.mat.emissionColor);
            HSL.e[2] *= 1.20; // luminosité
            HSL.e[1] *= 1.20; // saturation (valeurs subjectives)
            color newCol = hsl_to_rgb(HSL);
            return newCol;
        }
    }
    else return BLACK;
    
    color incomingLight = BLACK;
    color rayColor = WHITE;

    for (int i = 0; i<nbRebondMax; i++){

        HitInfo hitInfo = closest_hit(r, spheres, nbSpheres);

        if (hitInfo.didHit){
            r.origin = hitInfo.hitPoint;
            r.dir = vec3_normalize(add(hitInfo.normal,random_dir_no_norm(globalState, ind))); // sebastian lague

            material mat = hitInfo.mat;
            color emittedLight = multiply_scalar(mat.emissionColor, mat.emissionStrength);

            incomingLight = add(incomingLight,multiply(emittedLight, rayColor));
            rayColor = multiply(mat.diffuseColor, rayColor);
        }
        else{
            break;
        }
    }
    return incomingLight;
}

__global__ void render_canva(color* canva, int largeur_image, int hauteur_image, int nbRayonParPixel, int nbRebondMax, camera cam, hiprandState* states, sphere* spheres, int nbSpheres) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int ind = j * gridDim.x * blockDim.x + i;

    if (i < largeur_image && j < hauteur_image){
        int pixel_index = j*largeur_image+i;
        color totalLight = BLACK;
        
        for (int k=0; k<nbRayonParPixel; ++k){
            
            double u = ((double)i + randomDouble(states, ind, -0.5, 0.5))/(largeur_image-1);
            double v = ((double)j + randomDouble(states, ind, -0.5, 0.5))/(hauteur_image-1);

            ray r = get_ray(u, v, cam);
            totalLight = add(totalLight, tracer(r, nbRebondMax, states, ind, spheres, nbSpheres));
        }

        canva[pixel_index] = write_color_canva(totalLight, nbRayonParPixel);
    }
}

int main(){

    // CONSTANTES (paramètres de rendu)
    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////

    //format du fichier
    double ratio = 4.0 / 3.0;
    int largeur_image = 1000;
    int hauteur_image = (int)(largeur_image / ratio);

    //position de la camera
    double vfov = 110; // fov vertical en degrée
    point3 origin = {{-0.9, 0.9, -3.8}}; // position de la camera
    point3 target = {{0.2, 0, -2.8}}; // cible de la camera
    vec3 up = {{0, 1, 0.2}}; // permet de modifier la rotation selon l'axe z ({{0, 1, 0}} pour horizontal)

    //qualité et performance
    int nbRayonParPixel = 2000;
    int nbRebondMax = 5;
    
    int nbThreadsX = 8; // peut dépendre des GPU
    int nbThreadsY = 8; 

    bool useDenoiser = false;

    //position des sphères dans la scène
    sphere h_sphere_list[10] = {
        //{position du centre x, y, z}, rayon, {couleur de l'objet, couleur d'emission, force d'emission}
        {{{-501,0,0}}, 500, {GREEN, BLACK, 0.0}},                 
        {{{0,-501,0}}, 500, {WHITE, BLACK, 0.0}},                 
        {{{501, 0, 0}}, 500, {RED, BLACK, 0.0}},                  
        {{{-0.5, 1.4, -3}}, 0.5, {BLACK, {{1.0, 0.6, 0.2}}, 8.0}},   
        {{{0.5, 1.4, -3}}, 0.5, {BLACK, {{0.7, 0.2, 1.0}}, 8.0}},   
        {{{-0.5, -1.4, -3}}, 0.5, {BLACK, {{0.55, 0.863, 1.0}}, 5.0}},   
        {{{0.5, -1.4, -3}}, 0.5, {BLACK, {{0.431, 1.0, 0.596}}, 5.0}},   
        {{{0, 0, -504}}, 500, {WHITE, BLACK, 0.0}},               
        {{{0, 501, 0}}, 500, {WHITE, BLACK, 0.0}},                
        {{{0, 0, -3}}, 0.5, {SKY, BLACK, 0.0}}                    
    };

    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////

    int nbSpheres = sizeof(h_sphere_list) / sizeof(h_sphere_list[0]);

    // temps d'execution
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // nom du fichier
    char nomFichier[100];
    time_t maintenant = time(NULL); // Obtenir l'heure actuelle
    struct tm *temps = localtime(&maintenant); // Convertir en structure tm

    sprintf(nomFichier, "cam_rotation_%dRAYS_%dRB_%02d-%02d_%02dh%02d.ppm", nbRayonParPixel, nbRebondMax-1, temps->tm_mday, temps->tm_mon + 1, temps->tm_hour, temps->tm_min);

    FILE *fichier = fopen(nomFichier, "w");

    // camera
    camera cam = init_camera(origin, target, up, vfov, ratio);

    // tableau pour avoir chaque valeur de pixel au bon endroit (multithread et CUDA du coup)
    color* canva = (color*)malloc((largeur_image * hauteur_image)*sizeof(color));
    for (int i = 0; i < largeur_image*hauteur_image; i++) {
        canva[i] = BLACK;
    }

    // défini la taille des blocks et threads
    dim3 blocks(largeur_image/nbThreadsX+1, hauteur_image/nbThreadsY+1);
    dim3 threads(nbThreadsX, nbThreadsY);

    // alloue la mémoire pour states sur le device (gpu) (necessaire à la fontion de random)
    hiprandState* states;
    hipMalloc((void**) &states, (largeur_image * hauteur_image) * sizeof(hiprandState));

    // alloue la mémoire pour canva sur le device (gpu)
    color* canva_device;
    hipMalloc((void**)&canva_device, (largeur_image * hauteur_image)*sizeof(color));

    // alloue la mémoire pour d_sphere_list sur le device puis copie h_sphere_list (host) vers le device, optimisation
    sphere* d_sphere_list;
    hipMalloc((void**)&d_sphere_list, nbSpheres*sizeof(sphere));
    hipMemcpy(d_sphere_list, h_sphere_list, nbSpheres*sizeof(sphere), hipMemcpyHostToDevice);

    // initialise les "states" pour la fonction de random
    init_curand_state<<<blocks, threads>>>(states, largeur_image, hauteur_image);

    // lance le rendu de canva
    render_canva<<<blocks, threads>>>(canva_device, largeur_image, hauteur_image, nbRayonParPixel, nbRebondMax, cam, states, d_sphere_list, nbSpheres);

    // copie canva du device (gpu) vers l'host (cpu), puis free la mémoire de canva sur device
    hipMemcpy(canva, canva_device, (largeur_image * hauteur_image)*sizeof(color), hipMemcpyDeviceToHost);
    hipFree(canva_device);

    // utilise le denoiser si l'option est activée
    if (useDenoiser){
        denoiser(largeur_image, hauteur_image, canva, cam, h_sphere_list, nbSpheres);
    }
    
    //base_ppm et canva_to_ppm réecrit ici pour contrer l'appel de fprintf impossible depuis une fonction __host__ __device__
    fprintf(fichier, "P3\n%d %d\n255\n", largeur_image, hauteur_image);

    for (int j = hauteur_image-1; j >= 0  ; j--){ 
        for (int i = 0; i < largeur_image; i++){
            fprintf(fichier, "%d %d %d\n", (int)canva[j*largeur_image+i].e[0], (int)canva[j*largeur_image+i].e[1], (int)canva[j*largeur_image+i].e[2]);
        }
    }
    
    fclose(fichier);
    hipFree(d_sphere_list);
    hipFree(states);

    // enregistrer le moment d'arrivée
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    int minutes = (int)(elapsedTime / 60000);
    int seconds = (int)((elapsedTime - minutes * 60000) / 1000);
    
    fprintf(stderr, "\nFini.\n");
    fprintf(stderr, "Temps de rendu : %d min %d sec\n", minutes, seconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check les erreurs CUDA
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
    }

	return 0;
}
